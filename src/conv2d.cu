#include "hip/hip_runtime.h"
// how to increase the stack size if needed
// g++ -Wl,--stack,16777216 program.cpp -o program
// g++ -fsanitize=address -g program.cpp -o program
// g++ -fsanitize=address,undefined,leak -g program.cpp

// nvcc -std=c++20 -arch=sm_89 conv2d.cu && ./a.out

#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <vector>
#include <string>
#include "conv2d.hpp"
#include "device_ops.hpp"

template<typename Operation>
__global__ void convolve2d_kernel(const FLOAT *input_device, const FLOAT *kernel_device, FLOAT *output_device, Conv2DParams p, ImgProperty input_prop, ImgProperty output_prop, Operation op)
{
    /* Parameter documentation:


    */
    int idx_co = threadIdx.x + blockIdx.x * blockDim.x; // channel output
    int out_row = threadIdx.y + blockIdx.y * blockDim.y;
    int out_col = threadIdx.z + blockIdx.z * blockDim.z;

    if (out_row < output_prop.height && out_col < output_prop.width && idx_co < p.co)
    {
        FLOAT sum = 0.0f;

        // once padded, the first operation that will happen is on this particular index in the imaginary padded input (implicit)
        int in_row_start = out_row * p.s1 - p.p1;
        int in_col_start = out_col * p.s2 - p.p2;

        for (int idx_ci = 0; idx_ci < p.ci; idx_ci++)
        {
            for (int kernel_row = 0; kernel_row < p.k1; kernel_row++)
            {
                for (int kernel_col = 0; kernel_col < p.k2; kernel_col++)
                {
                    // co x ci x k1 x k2
                    FLOAT kernel_value = kernel_device[idx_co * (p.ci * p.k1 * p.k2) + idx_ci * (p.k1 * p.k2) + kernel_row * (p.k2) + kernel_col];

                    FLOAT input_value = ((in_row_start + kernel_row) >= 0 && (in_row_start + kernel_row) < input_prop.height &&
                                         (in_col_start + kernel_col) >= 0 && (in_col_start + kernel_col) < input_prop.width)
                                            ? input_device[idx_ci * input_prop.height * input_prop.width + (in_row_start + kernel_row) * input_prop.width + (in_col_start + kernel_col)]
                                            : 0.0f;

                    sum += input_value * kernel_value;
                }
            }
        }

        int o_index = idx_co * output_prop.height * output_prop.width + out_row * output_prop.width + out_col;
        output_device[o_index] = op.forward(sum);

    }
}

Conv2D::Conv2D(ImgProperty input_prop_, Conv2DParams params_)
    : input_prop(input_prop_), params(params_)
{
    validate_params();
    output_prop.height = (input_prop.height + 2 * params.p1 - params.k1) / params.s1 + 1;
    output_prop.width = (input_prop.width + 2 * params.p2 - params.k2) / params.s2 + 1;

    std::vector<int> output_Shape = {params.co, output_prop.height, output_prop.width};
    output_device.alloc(output_Shape);

    std::vector<int> kernel_Shape = {params.co,  params.ci, params.k1, params.k2};
    kernel_device.alloc(kernel_Shape);
}

Conv2D::~Conv2D()
{
    
}

template<typename Operation>
const DevicePointer<FLOAT>& Conv2D::forward(DevicePointer<FLOAT>& input_device, Operation op)
{
    const int TC = 8;
    dim3 threadcount(TC, TC, TC);
    dim3 blocks((params.co + TC - 1) / TC,
                (output_prop.height + TC - 1) / TC,
                (output_prop.width + TC - 1) / TC);

    convolve2d_kernel<<<blocks, threadcount>>>(input_device.get(), kernel_device.get(), output_device.get(), params, input_prop, output_prop, op);
    hipDeviceSynchronize();

    return output_device;
}

const DevicePointer<FLOAT>& Conv2D::forward(DevicePointer<FLOAT>& input_device)
{
    return forward(input_device, Identity());
}

void Conv2D::set_kernel(const std::vector<FLOAT> &kernel_data)
{
    size_t expected_size = params.co * params.ci * params.k1 * params.k2;
    if (kernel_data.size() != expected_size)
    {
        throw std::invalid_argument("Kernel size mismatch: expected " + std::to_string(expected_size) + " weights, got " + std::to_string(kernel_data.size()));
    }
    hipMemcpy(kernel_device.get(), kernel_data.data(), expected_size * sizeof(FLOAT), hipMemcpyHostToDevice);
}

const DevicePointer<FLOAT>& Conv2D::get_output()
{
    return output_device;
}

Conv2DParams Conv2D::get_param() const
{
    return params;
}

ImgProperty Conv2D::get_output_spec() const
{
    return output_prop;
}

ImgProperty Conv2D::get_input_spec() const
{
    return input_prop;
}

void Conv2D::validate_params()
{
    if (params.k1 % 2 == 0)
    {
        throw std::invalid_argument("k1 must be odd");
    }
    if (params.k2 % 2 == 0)
    {
        throw std::invalid_argument("k2 must be odd");
    }
    if (params.s1 <= 0)
    {
        throw std::invalid_argument("s1 (stride height) must be positive");
    }
    if (params.s2 <= 0)
    {
        throw std::invalid_argument("s2 (stride width) must be positive");
    }
    if (params.p1 < 0)
    {
        throw std::invalid_argument("p1 (padding height) must be non-negative");
    }
    if (params.p2 < 0)
    {
        throw std::invalid_argument("p2 (padding width) must be non-negative");
    }
}

/*
void Conv2D::set_kernel(const mxArray* mx_kernel)
{
    if (!mxIsNumeric(mx_kernel) || mxIsComplex(mx_kernel)) {
        throw std::invalid_argument("Kernel must be a real numeric array");
    }
    
    if (!mxIsSingle(mx_kernel)) {
        throw std::invalid_argument("Kernel must be single precision (use single() in MATLAB)");
    }
    
    size_t num_elements = mxGetNumberOfElements(mx_kernel);
    size_t expected_size = params.co * params.ci * params.k1 * params.k2;
    
    if (num_elements != expected_size) {
        throw std::invalid_argument("Kernel size mismatch: expected " + 
                                   std::to_string(expected_size) + " weights, got " + 
                                   std::to_string(num_elements));
    }
    
    float* data = (float*)mxGetData(mx_kernel);
    hipMemcpy(kernel_device.get(), data, expected_size * sizeof(FLOAT), hipMemcpyHostToDevice);
}

*/

#ifdef ACTIVATE_MAIN
int main()
{
    ImgProperty input_prop = {40, 60};                    // height, width
    Conv2DParams conv_params = {1, 1, 3, 16, 2, 2, 1, 1}; // k1,k2,ci,co,s1,s2,p1,p2

    Conv2D convlayer(input_prop, conv_params);

    float *input_device;
    // Fix: use conv_params values instead of undefined variables
    hipMalloc(&input_device, conv_params.ci * input_prop.height * input_prop.width * sizeof(float));
    hipMemset(input_device, 0, conv_params.ci * input_prop.height * input_prop.width * sizeof(float));

    convlayer.forward(input_device);

    hipFree(input_device);
    return 0;
}
#endif