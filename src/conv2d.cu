// Simple convolve 2d (non batched)

/*
Requirements:
1. No batch needed
2. preserve the size of the input image
3. input is  ci x H1 x W1
4. output is co x H2 x W2
5. Parameter count is co x ci x k1 x k2

Pytorch saves the parameters for conv layers in this format according to claude. Verify this
[out_channels, in_channels, kernel_height, kernel_width]

k1 for row and k2 for column
*/

// how to increase the stack size if needed
// g++ -Wl,--stack,16777216 program.cpp -o program
// g++ -fsanitize=address -g program.cpp -o program
// g++ -fsanitize=address,undefined,leak -g program.cpp

// nvcc -std=c++20 -arch=sm_89 conv2d.cu && ./a.out

#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <vector>

using FLOAT = float;

struct Conv2DParams
{
    int k1, k2, ci, co;
    int s1, s2, p1, p2;
};

struct ImgProperty
{
    int height;
    int width;
};

__global__ void convolve2d_kernel(const FLOAT *input_device, const FLOAT *kernel_device, FLOAT *output_device, Conv2DParams p, ImgProperty input_prop, ImgProperty output_prop)
{
    /* Parameter documentation:


    */
    int idx_co = threadIdx.x + blockIdx.x * blockDim.x; // channel output
    int out_row = threadIdx.y + blockIdx.y * blockDim.y;
    int out_col = threadIdx.z + blockIdx.z * blockDim.z;

    if (out_row < output_prop.height && out_col < output_prop.width && idx_co < p.co)
    {
        FLOAT sum = 0.0f;

        // once padded, the first operation that will happen is on this particular index in the imaginary padded input (implicit)
        int in_row_start = out_row * p.s1 - p.p1;
        int in_col_start = out_col * p.s2 - p.p2;

        for (int idx_ci = 0; idx_ci < p.ci; idx_ci++)
        {
            for (int kernel_row = 0; kernel_row < p.k1; kernel_row++)
            {
                for (int kernel_col = 0; kernel_col < p.k2; kernel_col++)
                {
                    // co x ci x k1 x k2
                    FLOAT kernel_value = kernel_device[idx_co * (p.ci * p.k1 * p.k2) + idx_ci * (p.k1 * p.k2) + kernel_row * (p.k2) + kernel_col];

                    FLOAT input_value = ((in_row_start + kernel_row) >= 0 && (in_row_start + kernel_row) < input_prop.height &&
                                         (in_col_start + kernel_col) >= 0 && (in_col_start + kernel_col) < input_prop.width)
                                            ? input_device[idx_ci * input_prop.height * input_prop.width + (in_row_start + kernel_row) * input_prop.width + (in_col_start + kernel_col)]
                                            : 0.0f;

                    sum += input_value * kernel_value;
                }
            }
        }

        int o_index = idx_co * output_prop.height * output_prop.width + out_row * output_prop.width + out_col;
        output_device[o_index] = sum;
    }
}

class Convolve2D
{

private:
    FLOAT *kernel_device; // co, ci, k1, k2 order for cache optimality. Thats how pytorch is built too. optimized for row major operations
    FLOAT *output_device; // co x output_height x output_width

    Conv2DParams params;
    ImgProperty input_prop, output_prop;
    dim3 threadcount, blocks;

public:
    Convolve2D(ImgProperty input_prop_, Conv2DParams params_) 
        : input_prop(input_prop_), params(params_)
    {
        /*
        compute the params
        */

        output_prop.height = (input_prop.height + 2 * params.p1 - params.k1) / params.s1 + 1;
        output_prop.width  = (input_prop.width  + 2 * params.p2 - params.k2) / params.s2 + 1;

        /*
        Set the kernel launch configuration
        */
        const int TC = 8;
        threadcount = dim3(TC, TC, TC);
        blocks = dim3((params.co + TC - 1) / TC,
                      (output_prop.height + TC - 1) / TC,
                      (output_prop.width + TC - 1) / TC);

        /*
        Allocate the memory for output and kernel
        */
        hipMalloc(&output_device,   params.co * output_prop.height * output_prop.width * sizeof(FLOAT));
        hipMemset(output_device, 0, params.co * output_prop.height * output_prop.width * sizeof(FLOAT));
        hipMalloc(&kernel_device,   params.co * params.ci * params.k1 * params.k2 * sizeof(FLOAT));
    
    }

    ~Convolve2D()
    {
        if (output_device != nullptr)
            hipFree(output_device);
        if (kernel_device != nullptr)
            hipFree(kernel_device);
    }

    void forward(const FLOAT *input_device)
    {
        convolve2d_kernel<<<blocks, threadcount>>>(input_device, kernel_device, output_device, params, input_prop, output_prop);
        hipDeviceSynchronize();
    }

    void set_kernel(const std::vector<FLOAT>& kernel_data)
    {
        size_t expected_size = params.co * params.ci * params.k1 * params.k2;
        if (kernel_data.size() != expected_size) {
            printf("Error: Expected %zu weights, got %zu\n", expected_size, kernel_data.size());
            exit(1);
        }
        hipMemcpy(kernel_device, kernel_data.data(), expected_size * sizeof(FLOAT), hipMemcpyHostToDevice);
    }

    FLOAT *get_output()
    {
        return output_device;
    }

    Conv2DParams get_param()
    {
        return params;
    }

    void validate_params()
    {
        if (params.k1 % 2 == 0) {
            throw std::invalid_argument("k1 must be odd");
        }
        if (params.k2 % 2 == 0) {
            throw std::invalid_argument("k2 must be odd");
        }
        if (params.s1 <= 0) {
            throw std::invalid_argument("s1 (stride height) must be positive");
        }
        if (params.s2 <= 0) {
            throw std::invalid_argument("s2 (stride width) must be positive");
        }
        if (params.p1 < 0) {
            throw std::invalid_argument("p1 (padding height) must be non-negative");
        }
        if (params.p2 < 0) {
            throw std::invalid_argument("p2 (padding width) must be non-negative");
        }

    }
};

#ifdef ACTIVATE_MAIN
int main()
{
    ImgProperty input_prop = {40, 60};  // height, width
    Conv2DParams conv_params = {1, 1, 3, 16, 2, 2, 1, 1};  // k1,k2,ci,co,s1,s2,p1,p2
    
    Convolve2D convlayer(input_prop, conv_params); 

    float *input_device;
    cudaMalloc(&input_device, ci * height * width * sizeof(float));

    cudaMemset(input_device, 0, ci * height * width * sizeof(float));

    convlayer.forward(input_device);

    cudaFree(input_device);

    return 0;
}
#endif