#include "hip/hip_runtime.h"

// Copyright 2025 Manoj Velmurugan
// SPDX-License-Identifier: MIT

#include <hipcub/hipcub.hpp>
#include "normalize.hpp"

__global__ void compute_mean_kernel(float* sum, int size) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *sum = *sum / size;
    }
}

__global__ void compute_variance_sum_kernel(const float* input, const float* mean, float* var_sum, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float diff = input[idx] - *mean;
        atomicAdd(var_sum, diff * diff);
    }
}

__global__ void instance_norm_kernel(const float* input, float* output, const float* mean, const float* variance, int size, float eps = 1e-5f)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        float std_inv = rsqrtf(*variance + eps);
        output[idx] = (input[idx] - *mean) * std_inv;
    }
}

ImageNorm2D::ImageNorm2D(ImgProperty input_prop_, float eps_) : input_prop(input_prop_), eps(eps_), size(input_prop_.height * input_prop_.width)
{
    output_prop = {input_prop_.channels, input_prop_.height, input_prop_.width};

    std::vector<int> output_Shape = {output_prop.channels, output_prop.height, output_prop.width};
    output_device.alloc(output_Shape);
    setup_workspace();
}

ImageNorm2D::~ImageNorm2D() 
{
    cleanup();
}

void ImageNorm2D::setup_workspace() 
{
    // Determine CUB workspace size
    hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes, (float*)nullptr, (float*)nullptr, size);
    
    // Allocate workspace
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipMalloc(&d_sum_result, sizeof(float));
    hipMalloc(&d_var_result, sizeof(float));
}

void ImageNorm2D::cleanup() {
    if (d_temp_storage) { hipFree(d_temp_storage); d_temp_storage = nullptr; }
    if (d_sum_result) { hipFree(d_sum_result); d_sum_result = nullptr; }
    if (d_var_result) { hipFree(d_var_result); d_var_result = nullptr; }
}

DevicePointer<FLOAT>& ImageNorm2D::forward(const DevicePointer<FLOAT>& input_device) {
    // Reset variance sum
    hipMemset(d_var_result, 0, sizeof(float));
    
    // Compute sum using CUB
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, input_device.get(), d_sum_result, size);
    
    // Convert sum to mean
    compute_mean_kernel<<<1, 1>>>(d_sum_result, size);
    
    // Compute variance sum
    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);
    compute_variance_sum_kernel<<<grid, block>>>(input_device.get(), d_sum_result, d_var_result, size);
    
    // Convert variance sum to variance
    compute_mean_kernel<<<1, 1>>>(d_var_result, size);
    
    // Apply normalization
    instance_norm_kernel<<<grid, block>>>(input_device.get(), output_device.get(), d_sum_result, d_var_result, size, eps);
    
    return output_device;
}