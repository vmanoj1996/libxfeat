#include <hip/hip_runtime.h>
#include "primitives.hpp"

// Common functionality to reuse 
template<typename T>
DevicePointer<T>::DevicePointer()
{
    
}

template<typename T>
DevicePointer<T>::~DevicePointer()
{
    if(ptr) hipFree(ptr);
}

template<typename T>
T* DevicePointer<T>::get() const
{
    return T;
}

template<typename T>
void DevicePointer<T>::alloc(int total_dim)
{
    hipMalloc(&ptr,    total_dim*sizeof(T));
    hipMemset(&ptr, 0, total_dim*sizeof(T));
}