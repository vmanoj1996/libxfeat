#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "primitives.hpp"
#include <string>
#include <stdexcept>
#include "device_ops.hpp"

// Common functionality to reuse 
template<typename T>
DevicePointer<T>::DevicePointer(int total_dim)
{
    alloc(total_dim);
}

template<typename T>
DevicePointer<T>::DevicePointer(const std::vector<T> &input, std::vector<int> dims_)
{
    alloc(dims_);
    set_value(input);
}

template<typename T>
DevicePointer<T>::~DevicePointer()
{
    if(ptr) hipFree(ptr);
}

template<typename T>
T* DevicePointer<T>::get()
{
    return ptr;
}

template<typename T>
void DevicePointer<T>::alloc(std::vector<int> dims_)
{
    int total_dim = 1;
    for(auto dim: dims_)
    {
        total_dim *= dim;
    }
    dims = dims_;
    
    alloc(total_dim);
}

template<typename T>
void DevicePointer<T>::alloc(int total_dim)
{
    size = total_dim;
    if(dims.empty())
    {
        // If there is no multiple dims in vector, put the total dim instead
        dims.push_back(total_dim);
    }
    hipMalloc(&ptr,    total_dim*sizeof(T));
    hipMemset(&ptr, 0, total_dim*sizeof(T));
}

template<typename T>
void DevicePointer<T>::set_value(const std::vector<T> &input)
{
    if(ptr){
        hipMemcpy(ptr, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice);
    }
}


template<typename T>
std::vector<T> DevicePointer<T>::get_value() const
{
    std::vector<T> result(size);
    if(ptr && size > 0) {
        hipError_t err = hipMemcpy(result.data(), ptr, 
                                    size * sizeof(T), 
                                    hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            throw std::runtime_error("hipMemcpy failed: " + std::string(hipGetErrorString(err)));
        }
    }
    return result;
}

template<typename T>
std::vector<int> DevicePointer<T>::get_shape() const
{
    std::vector<int> result = dims;

    return result;
}


__global__ void init_BatchNormRelu_kernel(DeviceOp** p) 
{
    *p = new BatchNormRelu();
}

__global__ void setup_BatchNormRelu_kernel(DeviceOp* p, float* d_mean, float* d_var, int N) 
{
    BatchNormRelu* bn = static_cast<BatchNormRelu*>(p);
    bn->mean = d_mean;
    bn->var  = d_var;
    bn->N    = N;
}

__global__ void delete_op_kernel(DeviceOp** p) 
{
    if(*p)
    {
        (*p)->deleter();
        delete *p;
        *p = nullptr;
    }
}

DeviceOp* create_BatchNormRelu(const std::vector<float>& mean_host, 
                               const std::vector<float>& var_host) 
{
    // Validate input
    if (mean_host.size() != var_host.size()) {
        throw std::invalid_argument("Mean and var vectors must have same size");
    }
    
    int N = mean_host.size();
    
    // Allocate device memory for arrays
    float* d_mean;
    float* d_var;
    hipMalloc(&d_mean, N * sizeof(float));
    hipMalloc(&d_var,  N * sizeof(float));
    
    // Copy data to device
    hipMemcpy(d_mean, mean_host.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_var, var_host.data(), N * sizeof(float), hipMemcpyHostToDevice);
    
    // Just declare a device pointer - no allocation needed
    DeviceOp* d_post_op;
    
    // Create and setup the object (device new allocates the object)
    init_BatchNormRelu_kernel<<<1,1>>>(&d_post_op);
    setup_BatchNormRelu_kernel<<<1,1>>>(d_post_op, d_mean, d_var, N);
    hipDeviceSynchronize();
    
    return d_post_op;
}