#include <hip/hip_runtime.h>
#include "primitives.hpp"
#include <string>
#include <stdexcept>
#include "device_ops.hpp"

// Common functionality to reuse 
// template<typename T>
// DevicePointer<T>::DevicePointer(int total_dim)
// {
//     alloc(total_dim);
// }

template<typename T>
DevicePointer<T>::DevicePointer(const std::vector<T> &input, std::vector<int> dims_)
{
    alloc(dims_);
    set_value(input);
}

template<typename T>
DevicePointer<T>::DevicePointer(const DevicePointer<T> &input)
{
    // copy constructor
    alloc(input.dims);
    hipMemcpy(ptr, input.get(), input.size * sizeof(T), hipMemcpyDeviceToDevice);
}

template<typename T>
DevicePointer<T>::~DevicePointer()
{
    if(ptr) hipFree(ptr);
}

template<typename T>
T* DevicePointer<T>::get() const
{
    return ptr;
}

template<typename T>
void DevicePointer<T>::alloc(std::vector<int> dims_)
{
    int total_dim = 1;
    for(auto dim: dims_)
    {
        total_dim *= dim;
    }
    dims = dims_;
    
    alloc(total_dim);
}

template<typename T>
void DevicePointer<T>::alloc(int total_dim)
{
    size = total_dim;
    if(dims.empty())
    {
        // If there is no multiple dims in vector, put the total dim instead
        dims.push_back(total_dim);
    }

    if(ptr) 
    {
        throw std::runtime_error("ptr already allocated\n");
    }

    hipError_t result = hipMalloc(&ptr,    total_dim*sizeof(T));
    hipMemset(ptr, 0, total_dim*sizeof(T));

    if (result != hipSuccess) {
        std::string error_msg = "CUDA malloc failed: " + std::string(hipGetErrorString(result)) + 
                               "\nCall stack:\n" + boost::stacktrace::to_string(boost::stacktrace::stacktrace());
        throw std::runtime_error(error_msg);
    }
}

template<typename T>
void DevicePointer<T>::set_value(const std::vector<T> &input)
{
    if(ptr){
        hipMemcpy(ptr, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice);
    }
}


template<typename T>
std::vector<T> DevicePointer<T>::get_value() const
{
    std::vector<T> result(size);
    if(ptr && size > 0) {
        hipError_t err = hipMemcpy(result.data(), ptr, size * sizeof(T), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            throw std::runtime_error("hipMemcpy failed: " + std::string(hipGetErrorString(err)));
        }
    }
    return result;
}

template<typename T>
std::vector<int> DevicePointer<T>::get_shape() const
{
    std::vector<int> result = dims;

    return result;
}

template<typename T>
void DevicePointer<T>::print_shape() const
{
    for(auto dim: dims)
    {
        std::cout<<dim<<" ";
    }
    std::cout<<"\n";
}
