#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include "conv2d_gemm.hpp"

template<int K, int CI, int CO, int S, int P>
void profile_template(int h, int w, std::ofstream& csv, std::ofstream& detailed_csv) {
    constexpr Conv2DParams params{K, K, CI, CO, S, S, P, P};
    ImgProperty input_prop{CI, h, w};
    
    // Create layer
    std::vector<FLOAT> kernel(CO * CI * K * K, 0.1f);
    auto layer_ptr = conv2d<params>(input_prop, kernel, Identity(), 0);
    auto conv = static_cast<Conv2D<params, Identity>*>(layer_ptr.get());
    
    // Create input
    std::vector<FLOAT> input(CI * h * w, 1.0f);
    DevicePointer<FLOAT> input_d(input, {CI, h, w});
    
    // Test different thread configurations
    float best_time = 1e9;
    int best_tc1 = 2, best_tc2 = 32;
    
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    
    for(int tc1 = 1; tc1 <= 128; tc1 *= 2) {
        for(int tc2 = 1; tc2 <= 128; tc2 *= 2) {
            if(tc1 * tc2 > 1024) continue;
            
            // Warmup
            try 
            {
                for(int i = 0; i < 5; i++) 
                    conv->forward_profile(input_d, tc1, tc2);
            }
            catch(const std::exception& e)
            {
                // kernel launch failed with this config.
                continue;
            }
            
            // Time
            hipEventRecord(start);
            for(int i = 0; i < 100; i++) 
                conv->forward_profile(input_d, tc1, tc2);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            float avg_ms = ms / 100.0f;
            
            // Write to detailed CSV
            detailed_csv << K << "," << CI << "," << CO << "," << S << "," << P << ","
                        << h << "," << w << "," << tc1 << "," << tc2 << "," 
                        << avg_ms << std::endl;

            if(ms < best_time) {
                best_time = ms;
                best_tc1 = tc1;
                best_tc2 = tc2;
            }
        }
    }
    
    float avg_time = best_time / 100.0f;
    
    // Console output
    // std::cout << K << "x" << K << " " << CI << "->" << CO 
    //           << " s=" << S << " p=" << P << " @ " << h << "x" << w
    //           << ": " << avg_time << "ms (best: " 
    //           << best_tc1 << "x" << best_tc2 << ")" << std::endl;
    
    // CSV output (summary)
    csv << K << "," << CI << "," << CO << "," << S << "," << P << ","
        << h << "," << w << "," << avg_time << "," 
        << best_tc1 << "," << best_tc2 << std::endl;

    // best time in different format
    std::cout << "{" << K << ", " << CI << ", " << CO << ", " << S << ", " << P << ", " 
                        << h << ", " << w << ", " << best_tc1 << ", " << best_tc2 << "}," << std::endl;
    
    hipEventDestroy(start); hipEventDestroy(stop);
}

int main() {
    // Clear and create new CSV files
    std::ofstream csv("conv2d_profile.csv", std::ios::trunc);
    csv << "kernel,in_ch,out_ch,stride,pad,height,width,time_ms,best_tc1,best_tc2" << std::endl;
    
    std::ofstream detailed_csv("conv2d_profile_detailed.csv", std::ios::trunc);
    detailed_csv << "kernel,in_ch,out_ch,stride,pad,height,width,tc1,tc2,time_ms" << std::endl;
    
    std::cout << "Profiling Conv2D layers with thread tuning...\n" << std::endl;
    
    // All unique configurations from your file with actual dimensions
    profile_template<3, 1, 4, 1, 1>(480, 640, csv, detailed_csv);
    profile_template<3, 4, 8, 2, 1>(480, 640, csv, detailed_csv);
    profile_template<3, 8, 8, 1, 1>(240, 320, csv, detailed_csv);
    profile_template<3, 8, 24, 2, 1>(240, 320, csv, detailed_csv);
    profile_template<1, 1, 24, 1, 0>(120, 160, csv, detailed_csv);
    profile_template<3, 24, 24, 1, 1>(120, 160, csv, detailed_csv);
    profile_template<3, 24, 64, 2, 1>(120, 160, csv, detailed_csv);
    profile_template<3, 64, 64, 1, 1>(60, 80, csv, detailed_csv);
    profile_template<1, 64, 64, 1, 0>(60, 80, csv, detailed_csv);
    profile_template<3, 64, 64, 2, 1>(60, 80, csv, detailed_csv);
    profile_template<3, 64, 64, 1, 1>(30, 40, csv, detailed_csv);
    profile_template<3, 64, 128, 2, 1>(30, 40, csv, detailed_csv);
    profile_template<3, 128, 128, 1, 1>(15, 20, csv, detailed_csv);
    profile_template<1, 128, 64, 1, 0>(15, 20, csv, detailed_csv);
    profile_template<1, 64, 1, 1, 0>(60, 80, csv, detailed_csv);
    profile_template<1, 64, 65, 1, 0>(60, 80, csv, detailed_csv);
    
    csv.close();
    detailed_csv.close();
    std::cout << "\nResults saved to conv2d_profile.csv (summary) and conv2d_profile_detailed.csv (all configs)" << std::endl;
    
    return 0;
}